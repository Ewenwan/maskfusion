#include "hip/hip_runtime.h"
/*
 * This file is part of ElasticFusion.
 * 图像操作 常用函数实现
 * 高斯下采样、深度值三角变换 归一化map 深度转3D点 点转深度 拷贝 变形
 * 2dMAP 间隔 列数行存储 x，y，z值
 * 归一化map: 相邻 三点 构成 两向量 叉乘向量 再归一化
 *  Author: Anatoly Baskeheev, Itseez Ltd, (myname.mysurname@mycompany.com)
 */

#include "cudafuncs.cuh"
#include "convenience.cuh"
#include "operators.cuh"

// 金字塔高斯模糊下采样 函数核 
__global__ void pyrDownGaussKernel (const PtrStepSz<float> src, PtrStepSz<float> dst, float sigma_color)
{
    // 二维线程块
    int x = blockIdx.x * blockDim.x + threadIdx.x;// 按行索引的线程索引
    int y = blockIdx.y * blockDim.y + threadIdx.y;// 按列索引的线程索引

    if (x >= dst.cols || y >= dst.rows)
        return;

    const int D = 5;

    int center = src.ptr (2 * y)[2 * x];

    int x_mi = max(0, 2*x - D/2) - 2*x;
    int y_mi = max(0, 2*y - D/2) - 2*y;

    int x_ma = min(src.cols, 2*x -D/2+D) - 2*x;
    int y_ma = min(src.rows, 2*y -D/2+D) - 2*y;

    float sum = 0;
    float wall = 0;

    float weights[] = {0.375f, 0.25f, 0.0625f} ;

    for(int yi = y_mi; yi < y_ma; ++yi)
        for(int xi = x_mi; xi < x_ma; ++xi)
        {
            int val = src.ptr (2*y + yi)[2*x + xi];

            if (abs (val - center) < 3 * sigma_color)
            {
                sum += val * weights[abs(xi)] * weights[abs(yi)];
                wall += weights[abs(xi)] * weights[abs(yi)];
            }
        }


    dst.ptr (y)[x] = static_cast<int>(sum / wall);
}

// 金字塔下采样函数=============================================================================
void pyrDown(const DeviceArray2D<unsigned short> & src, DeviceArray2D<unsigned short> & dst)
{
    // src 目标图像      gpu二维数组
    // dst 下采样后的图像 
    dst.create (src.rows () / 2, src.cols () / 2);// 下采样后的尺寸
//  1. dim3是基亍uint3定义的矢量类型，相当亍由3个unsigned int型组成的结构体。
//       uint3类型有三个数据成员unsigned int x; unsigned int y; unsigned int z;
//  2. 可使用亍一维、二维或三维的索引来标识线程，构成一维、二维或三维线程块。
//  3. dim3结构类型变量用在核函数调用的<<<,>>>中。
// https://github.com/Ewenwan/ShiYanLou/blob/master/CUDA/readme.md
    
    dim3 block (32, 8);// 二维线程块
    // 线程格
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));

    const float sigma_color = 30;

    pyrDownGaussKernel<<<grid, block>>>(src, dst, sigma_color);
    cudaCheckError();
}

// 深度值 + 相机内参数计算 x,y,z三维点 核函数==========================================
// Generate a vertex map 'vmap' based on the depth map 'depth' and camera parameters
__global__ void computeVmapKernel(const PtrStepSz<float> depth, 
                                  PtrStep<float> vmap, 
                                  float fx_inv, 
                                  float fy_inv, 
                                  float cx, 
                                  float cy, 
                                  float depthCutoff)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;

    if(u < depth.cols && v < depth.rows)
    {
        float z = depth.ptr (v)[u];// 深度值=============

        if(z > 0.0f && z < depthCutoff)
        {
            float vx = z * (u - cx) * fx_inv; // x，y,z三坐标值
            float vy = z * (v - cy) * fy_inv;
            float vz = z;
            // 间隔 列数行存储 x，y，z值
            vmap.ptr (v                 )[u] = vx;// 第一行存 x
            vmap.ptr (v + depth.rows    )[u] = vy;// 第二行(与第一行隔开列数个行)存 y
            vmap.ptr (v + depth.rows * 2)[u] = vz;// 第三行(与第二行隔开列数个行)存 z
        }
        else
        {
            vmap.ptr (v)[u] = __int_as_float(0x7fffffff); /*HIP_NAN_F*/
            vmap.ptr (v + depth.rows * 2)[u] = 0;
        }
    }
}

// 深度值 + 相机内参数计算 x,y,z三维点 三角变换=================================================
void createVMap(const CameraModel& intr, 
                const DeviceArray2D<float> & depth, 
                DeviceArray2D<float> & vmap, 
                const float depthCutoff)
{
    vmap.create (depth.rows () * 3, depth.cols ());// 一行变3行，存x,y,z三个值

    dim3 block (32, 8);// 二维 线程块
    dim3 grid (1, 1, 1);// 二维线程 格
    grid.x = getGridDim (depth.cols (), block.x);
    grid.y = getGridDim (depth.rows (), block.y);
    // 相机内参数
    float fx = intr.fx, cx = intr.cx;
    float fy = intr.fy, cy = intr.cy;

    computeVmapKernel<<<grid, block>>>(depth, vmap, 1.f / fx, 1.f / fy, cx, cy, depthCutoff);
    cudaSafeCall (hipGetLastError ());
}

// 归一化map=====相邻三点构成两向量叉乘向量再归一化==========
__global__ void computeNmapKernel(int rows, int cols, const PtrStep<float> vmap, PtrStep<float> nmap)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;

    if (u >= cols || v >= rows)
        return;

    if (u == cols - 1 || v == rows - 1)
    {
        nmap.ptr (v)[u] = __int_as_float(0x7fffffff); /*HIP_NAN_F*/
        return;
    }

    float3 v00, v01, v10;
    // x
    v00.x = vmap.ptr (v  )[u];      //  [V,U]  [V,U+1]
    v01.x = vmap.ptr (v  )[u + 1];  //  [V+1,U] 
    v10.x = vmap.ptr (v + 1)[u];

    if (!isnan (v00.x) && !isnan (v01.x) && !isnan (v10.x))// 右边点，下边点的x值都存在(不是噪点...)
    {
        // 对应隔列书行存储的为 y值
        v00.y = vmap.ptr (v + rows)[u];
        v01.y = vmap.ptr (v + rows)[u + 1];// 右边点
        v10.y = vmap.ptr (v + 1 + rows)[u];// 下边点
        
        // z
        v00.z = vmap.ptr (v + 2 * rows)[u];
        v01.z = vmap.ptr (v + 2 * rows)[u + 1];
        v10.z = vmap.ptr (v + 1 + 2 * rows)[u];

        float3 r = normalized (cross (v01 - v00, v10 - v00));// 相邻 三点 构成 两向量 叉乘向量 再归一化

        nmap.ptr (v       )[u] = r.x;
        nmap.ptr (v + rows)[u] = r.y;
        nmap.ptr (v + 2 * rows)[u] = r.z;
    }
    else
        nmap.ptr (v)[u] = __int_as_float(0x7fffffff); /*HIP_NAN_F*/
}
// 归一化map===============
void createNMap(const DeviceArray2D<float>& vmap, 
                DeviceArray2D<float>& nmap)
{
    nmap.create (vmap.rows (), vmap.cols ());

    int rows = vmap.rows () / 3;// 输入的map的 行数是扩大了3倍的，一行深度变3行X，Y，Z
    int cols = vmap.cols ();

    dim3 block (32, 8);
    dim3 grid (1, 1, 1);
    grid.x = getGridDim (cols, block.x);
    grid.y = getGridDim (rows, block.y);

    computeNmapKernel<<<grid, block>>>(rows, cols, vmap, nmap);
    cudaSafeCall (hipGetLastError ());
}




__global__ void tranformMapsKernel(int rows, int cols, 
                                   const PtrStep<float> vmap_src, 
                                   const PtrStep<float> nmap_src,
                                   const mat33 Rmat, 
                                   const float3 tvec, PtrStepSz<float> vmap_dst,
                                   PtrStep<float> nmap_dst)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < cols && y < rows)
    {
        //vertexes
        float3 vsrc, vdst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
        vsrc.x = vmap_src.ptr (y)[x];

        if (!isnan (vsrc.x))
        {
            vsrc.y = vmap_src.ptr (y + rows)[x];
            vsrc.z = vmap_src.ptr (y + 2 * rows)[x];

            vdst = Rmat * vsrc + tvec;// 3D点刚体变换  

            vmap_dst.ptr (y + rows)[x] = vdst.y;
            vmap_dst.ptr (y + 2 * rows)[x] = vdst.z;
        }

        vmap_dst.ptr (y)[x] = vdst.x;

        //normals
        float3 nsrc, ndst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
        nsrc.x = nmap_src.ptr (y)[x];

        if (!isnan (nsrc.x))
        {
            nsrc.y = nmap_src.ptr (y + rows)[x];
            nsrc.z = nmap_src.ptr (y + 2 * rows)[x];

            ndst = Rmat * nsrc;// 归一化MAP只需要 旋转变换即可！！！！！！！！

            nmap_dst.ptr (y + rows)[x] = ndst.y;
            nmap_dst.ptr (y + 2 * rows)[x] = ndst.z;
        }

        nmap_dst.ptr (y)[x] = ndst.x;
    }
}
// 3Di点 RT刚体变换  ===========================================
void tranformMaps(const DeviceArray2D<float>& vmap_src,
                  const DeviceArray2D<float>& nmap_src,
                  const mat33& Rmat, const float3& tvec,
                  DeviceArray2D<float>& vmap_dst, DeviceArray2D<float>& nmap_dst)
{
    int cols = vmap_src.cols();
    int rows = vmap_src.rows() / 3;

    vmap_dst.create(rows * 3, cols);
    nmap_dst.create(rows * 3, cols);

    dim3 block(32, 8);
    dim3 grid(1, 1, 1);
    grid.x = getGridDim(cols, block.x);
    grid.y = getGridDim(rows, block.y);

    tranformMapsKernel<<<grid, block>>>(rows, cols, vmap_src, nmap_src, Rmat, tvec, vmap_dst, nmap_dst);
    cudaSafeCall(hipGetLastError());
}



// 拷贝 3D地图点和归一化MAP===========================
__global__ void copyMapsKernel(int rows, int cols, 
                               const float * vmap_src, 
                               const float * nmap_src,
                               PtrStepSz<float> vmap_dst, 
                               PtrStep<float> nmap_dst)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < cols && y < rows)
    {
        //vertexes
        float3 vsrc, vdst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

        vsrc.x = vmap_src[y * cols * 4 + (x * 4) + 0];
        vsrc.y = vmap_src[y * cols * 4 + (x * 4) + 1];
        vsrc.z = vmap_src[y * cols * 4 + (x * 4) + 2];

        if(!(vsrc.z == 0))
        {
            vdst = vsrc;
        }

        vmap_dst.ptr (y)[x] = vdst.x;
        vmap_dst.ptr (y + rows)[x] = vdst.y;
        vmap_dst.ptr (y + 2 * rows)[x] = vdst.z;

        //normals
        float3 nsrc, ndst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

        nsrc.x = nmap_src[y * cols * 4 + (x * 4) + 0];
        nsrc.y = nmap_src[y * cols * 4 + (x * 4) + 1];
        nsrc.z = nmap_src[y * cols * 4 + (x * 4) + 2];

        if(!(vsrc.z == 0))
        {
            ndst = nsrc;
        }

        nmap_dst.ptr (y)[x] = ndst.x;
        nmap_dst.ptr (y + rows)[x] = ndst.y;
        nmap_dst.ptr (y + 2 * rows)[x] = ndst.z;
    }
}
// 拷贝 3D地图点和归一化MAP========================================
void copyMaps(const DeviceArray<float>& vmap_src,
              const DeviceArray<float>& nmap_src,
              DeviceArray2D<float>& vmap_dst,
              DeviceArray2D<float>& nmap_dst)
{
    int cols = vmap_dst.cols();
    int rows = vmap_dst.rows() / 3;

    vmap_dst.create(rows * 3, cols);
    nmap_dst.create(rows * 3, cols);

    dim3 block(32, 8);
    dim3 grid(1, 1, 1);
    grid.x = getGridDim(cols, block.x);
    grid.y = getGridDim(rows, block.y);

    copyMapsKernel<<<grid, block>>>(rows, cols, vmap_src, nmap_src, vmap_dst, nmap_dst);
    cudaSafeCall(hipGetLastError());
}



// 自定义高斯核下采样====================================
__global__ void pyrDownKernelGaussF(const PtrStepSz<float> src, 
                                    PtrStepSz<float> dst, 
                                    float * gaussKernel)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst.cols || y >= dst.rows)
        return;

    const int D = 5;

    float center = src.ptr (2 * y)[2 * x];

    int tx = min (2 * x - D / 2 + D, src.cols - 1);
    int ty = min (2 * y - D / 2 + D, src.rows - 1);
    int cy = max (0, 2 * y - D / 2);

    float sum = 0;
    int count = 0;

    for (; cy < ty; ++cy)
    {
        for (int cx = max (0, 2 * x - D / 2); cx < tx; ++cx)
        {
            if(!isnan(src.ptr (cy)[cx]))
            {
                sum += src.ptr (cy)[cx] * gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
                count += gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
            }
        }
    }
    dst.ptr (y)[x] = (float)(sum / (float)count);
}
// 5×5高斯核下采样==float类型======================
void pyrDownGaussF(const DeviceArray2D<float>& src, 
                   DeviceArray2D<float> & dst)
{
    dst.create (src.rows () / 2, src.cols () / 2);

    dim3 block (32, 8);
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));

    const float gaussKernel[25] = 
                   {1, 4, 6, 4, 1,
                    4, 16, 24, 16, 4,
                    6, 24, 36, 24, 6,
                    4, 16, 24, 16, 4,
                    1, 4, 6, 4, 1};

    float * gauss_cuda;

    cudaSafeCall(hipMalloc((void**) &gauss_cuda, sizeof(float) * 25));
    cudaSafeCall(hipMemcpy(gauss_cuda, &gaussKernel[0], sizeof(float) * 25, hipMemcpyHostToDevice));

    pyrDownKernelGaussF<<<grid, block>>>(src, dst, gauss_cuda);
    cudaCheckError();

    hipFree(gauss_cuda);
}


// map变形==================================
template<bool normalize>
__global__ void resizeMapKernel(int drows, int dcols, int srows, 
                                const PtrStep<float> input, 
                                PtrStep<float> output)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= dcols || y >= drows)
        return;

    const float qnan = __int_as_float(0x7fffffff);

    int xs = x * 2;
    int ys = y * 2;

    float x00 = input.ptr (ys + 0)[xs + 0];
    float x01 = input.ptr (ys + 0)[xs + 1];
    float x10 = input.ptr (ys + 1)[xs + 0];
    float x11 = input.ptr (ys + 1)[xs + 1];

    if (isnan (x00) || isnan (x01) || isnan (x10) || isnan (x11))
    {
        output.ptr (y)[x] = qnan;
        return;
    }
    else
    {
        float3 n;

        n.x = (x00 + x01 + x10 + x11) / 4;

        float y00 = input.ptr (ys + srows + 0)[xs + 0];
        float y01 = input.ptr (ys + srows + 0)[xs + 1];
        float y10 = input.ptr (ys + srows + 1)[xs + 0];
        float y11 = input.ptr (ys + srows + 1)[xs + 1];

        n.y = (y00 + y01 + y10 + y11) / 4;

        float z00 = input.ptr (ys + 2 * srows + 0)[xs + 0];
        float z01 = input.ptr (ys + 2 * srows + 0)[xs + 1];
        float z10 = input.ptr (ys + 2 * srows + 1)[xs + 0];
        float z11 = input.ptr (ys + 2 * srows + 1)[xs + 1];

        n.z = (z00 + z01 + z10 + z11) / 4;

        if (normalize)
            n = normalized (n);

        output.ptr (y        )[x] = n.x;
        output.ptr (y + drows)[x] = n.y;
        output.ptr (y + 2 * drows)[x] = n.z;
    }
}
// map变形==================================
template<bool normalize>
void resizeMap(const DeviceArray2D<float>& input, DeviceArray2D<float>& output)
{
    int in_cols = input.cols ();
    int in_rows = input.rows () / 3;

    int out_cols = in_cols / 2;
    int out_rows = in_rows / 2;

    output.create (out_rows * 3, out_cols);

    dim3 block (32, 8);
    dim3 grid (getGridDim (out_cols, block.x), getGridDim (out_rows, block.y));
    resizeMapKernel<normalize><< < grid, block>>>(out_rows, out_cols, in_rows, input, output);
    cudaCheckError();
    cudaSafeCall (hipDeviceSynchronize ());
}

void resizeVMap(const DeviceArray2D<float>& input, DeviceArray2D<float>& output)
{
    resizeMap<false>(input, output);
}

void resizeNMap(const DeviceArray2D<float>& input, DeviceArray2D<float>& output)
{
    resizeMap<true>(input, output);
}

//FIXME Remove
/*
void launch_kernel(float4 *pos, unsigned int mesh_width,
                   unsigned int mesh_height, float time)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    simple_vbo_kernel<<< grid, block>>>(pos, mesh_width, mesh_height, time);
}*/

//FIXME Remove
/*
__global__ void testKernel(hipSurfaceObject_t tex)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= 960 || y >= 540)
        return;

    / *
    const int D = 5;

    float center = src.ptr (2 * y)[2 * x];

    int tx = min (2 * x - D / 2 + D, src.cols - 1);
    int ty = min (2 * y - D / 2 + D, src.rows - 1);
    int cy = max (0, 2 * y - D / 2);

    float sum = 0;
    int count = 0;

    for (; cy < ty; ++cy)
    {
        for (int cx = max (0, 2 * x - D / 2); cx < tx; ++cx)
        {
            if(!isnan(src.ptr (cy)[cx]))
            {
                sum += src.ptr (cy)[cx] * gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
                count += gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
            }
        }
    }* /
    //dst.ptr (y)[x] = (float)(sum / (float)count);
    //data[y * 960 + x] = x / 960.0;
    //data[8] = 0.4;
    float1 test = make_float1(0.99);
    surf2Dwrite(test, tex, x*sizeof(float1), y);
}

//FIXME Remove
void testCuda(hipSurfaceObject_t surface)//(float* data)
{
    //dst.create (src.rows () / 2, src.cols () / 2);

    dim3 block (32, 8);
    dim3 grid (getGridDim (960, block.x), getGridDim (540, block.y));

    testKernel<<<grid, block>>>(surface);
    cudaCheckError();
}*/


// 5×5高斯核下采样==unsigned char类型======================
__global__ void pyrDownKernelIntensityGauss(const PtrStepSz<unsigned char> src, 
                                            PtrStepSz<unsigned char> dst, 
                                            float * gaussKernel)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst.cols || y >= dst.rows)
        return;

    const int D = 5;

    int center = src.ptr (2 * y)[2 * x];

    int tx = min (2 * x - D / 2 + D, src.cols - 1);
    int ty = min (2 * y - D / 2 + D, src.rows - 1);
    int cy = max (0, 2 * y - D / 2);

    float sum = 0;
    int count = 0;

    for (; cy < ty; ++cy)
        for (int cx = max (0, 2 * x - D / 2); cx < tx; ++cx)
        {
            //This might not be right, but it stops incomplete model images from making up colors
            if(src.ptr (cy)[cx] > 0)
            {
                sum += src.ptr (cy)[cx] * gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
                count += gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
            }
        }
    dst.ptr (y)[x] = (sum / (float)count);
}
// 5×5高斯核下采样==unsigned char类型======================
void pyrDownUcharGauss(const DeviceArray2D<unsigned char>& src, 
                       DeviceArray2D<unsigned char> & dst)
{
    dst.create (src.rows () / 2, src.cols () / 2);

    dim3 block (32, 8);
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));
    
    //  5×5高斯核
    const float gaussKernel[25] = {1, 4, 6, 4, 1,
                    4, 16, 24, 16, 4,
                    6, 24, 36, 24, 6,
                    4, 16, 24, 16, 4,
                    1, 4, 6, 4, 1};

    float * gauss_cuda;

    hipMalloc((void**) &gauss_cuda, sizeof(float) * 25);
    hipMemcpy(gauss_cuda, &gaussKernel[0], sizeof(float) * 25, hipMemcpyHostToDevice);

    pyrDownKernelIntensityGauss<<<grid, block>>>(src, dst, gauss_cuda);
    cudaCheckError();

    hipFree(gauss_cuda);
}

/*void pyrDown2(const DeviceArray2D<unsigned char> & src, DeviceArray2D<unsigned char> & dst)
{
    dst.create (src.rows () / 2, src.cols () / 2);

    dim3 block (32, 8);
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));

    //pyrDownUcharGauss<<<grid, block>>>(src, dst);
    pyrDownUcharGauss()
    cudaCheckError();
}*/

// 3d点转 深度图===================================
__global__ void verticesToDepthKernel(const float * vmap_src, PtrStepSz<float> dst, float cutOff)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst.cols || y >= dst.rows)
        return;

    float z = vmap_src[y * dst.cols * 4 + (x * 4) + 2];

    dst.ptr(y)[x] = z > cutOff || z <= 0 ? __int_as_float(0x7fffffff)/*HIP_NAN_F*/ : z;
}
// 3d map转 深度图===================================
void verticesToDepth(DeviceArray<float>& vmap_src, DeviceArray2D<float> & dst, float cutOff)
{
    dim3 block (32, 8);
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));

    verticesToDepthKernel<<<grid, block>>>(vmap_src, dst, cutOff);
    cudaCheckError();
}

texture<uchar4, 2, hipReadModeElementType> inTex;

// RGB图转灰度图=======================================
__global__ void bgr2IntensityKernel(PtrStepSz<unsigned char> dst)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst.cols || y >= dst.rows)
        return;

    uchar4 src = tex2D(inTex, x, y);
//  Gray = R*0.299 + G*0.587 + B*0.114; 
//  Gray = (R*299 + G*587 + B*114 + 500); // 1000所以需要加上500来实现四舍五入。
//  Gray = (R*30 + G*59 + B*11 + 50) / 100; 
//  Gray = (R*38 + G*75 + B*15) >> 7
//  Gray = (R + (WORD)G<<1 + B) >> 2
    int value = (float)src.x * 0.114f + (float)src.y * 0.299f + (float)src.z * 0.587f;

    dst.ptr (y)[x] = value;
}
// RGB图转灰度图=======================================
void imageBGRToIntensity(hipArray * cuArr, DeviceArray2D<unsigned char> & dst)
{
    dim3 block (32, 8);
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));

    cudaSafeCall(hipBindTextureToArray(inTex, cuArr));

    bgr2IntensityKernel<<<grid, block>>>(dst);

    cudaCheckError();

    cudaSafeCall(hipUnbindTexture(inTex));
}

__constant__ float gsobel_x3x3[9];
__constant__ float gsobel_y3x3[9];

__global__ void applyKernel(const PtrStepSz<unsigned char> src, 
                            PtrStep<short> dx, 
                            PtrStep<short> dy)
{

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if(x >= src.cols || y >= src.rows)
    return;

  float dxVal = 0;
  float dyVal = 0;

  int kernelIndex = 8;
  for(int j = max(y - 1, 0); j <= min(y + 1, src.rows - 1); j++)
  {
      for(int i = max(x - 1, 0); i <= min(x + 1, src.cols - 1); i++)
      {
          dxVal += (float)src.ptr(j)[i] * gsobel_x3x3[kernelIndex];
          dyVal += (float)src.ptr(j)[i] * gsobel_y3x3[kernelIndex];
          --kernelIndex;
      }
  }

  dx.ptr(y)[x] = dxVal;
  dy.ptr(y)[x] = dyVal;
}

// 计算图像梯度====================================
void computeDerivativeImages(DeviceArray2D<unsigned char>& src, DeviceArray2D<short>& dx, DeviceArray2D<short>& dy)
{
    static bool once = false;

    if(!once)
    {
        // x 水平方向 导数模板 
        float gsx3x3[9] = {0.52201,  0.00000, -0.52201,
                           0.79451, -0.00000, -0.79451,
                           0.52201,  0.00000, -0.52201};
        // y 垂直方向 导数模板
        float gsy3x3[9] = {0.52201, 0.79451, 0.52201,
                           0.00000, 0.00000, 0.00000,
                           -0.52201, -0.79451, -0.52201};

        hipMemcpyToSymbol(HIP_SYMBOL(gsobel_x3x3), gsx3x3, sizeof(float) * 9);
        hipMemcpyToSymbol(HIP_SYMBOL(gsobel_y3x3), gsy3x3, sizeof(float) * 9);

        cudaSafeCall(hipGetLastError());
        cudaSafeCall(hipDeviceSynchronize());

        once = true;
    }

    dim3 block(32, 8);
    dim3 grid(getGridDim (src.cols (), block.x), getGridDim (src.rows (), block.y));

    applyKernel<<<grid, block>>>(src, dx, dy);

    cudaSafeCall(hipGetLastError());
    cudaSafeCall(hipDeviceSynchronize());
}


// 
__global__ void projectPointsKernel(const PtrStepSz<float> depth,
                                    PtrStepSz<float3> cloud,
                                    const float invFx,
                                    const float invFy,
                                    const float cx,
                                    const float cy)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= depth.cols || y >= depth.rows)
        return;

    float z = depth.ptr(y)[x];

    cloud.ptr(y)[x].x = (float)((x - cx) * z * invFx);
    cloud.ptr(y)[x].y = (float)((y - cy) * z * invFy);
    cloud.ptr(y)[x].z = z;
}

// 深度图 转 点云， 维度不变，每个元素是一个三维变量，分别存储x，y，z
void projectToPointCloud(const DeviceArray2D<float> & depth,
                         const DeviceArray2D<float3> & cloud,
                         CameraModel & intrinsics,
                         const int & level)
{
    dim3 block (32, 8);
    dim3 grid (getGridDim (depth.cols (), block.x), getGridDim (depth.rows (), block.y));

    CameraModel intrinsicsLevel = intrinsics(level);

    projectPointsKernel<<<grid, block>>>(depth, cloud, 1.0f / intrinsicsLevel.fx, 1.0f / intrinsicsLevel.fy, intrinsicsLevel.cx, intrinsicsLevel.cy);
    cudaCheckError();
    cudaSafeCall (hipDeviceSynchronize ());
}
